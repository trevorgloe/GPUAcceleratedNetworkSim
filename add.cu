
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

__device__ float e_taylor(float x, int deg);
__device__ int factorial(int n);

// Kernel function to add the elements of two arrays
__global__
void add(int n, float *x, float *y)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < n; i += stride)
    y[i] = e_taylor(y[i], 20);
}

__device__
float e_taylor(float x, int deg){
  float ans = 0.0f;
  for (int i = 0; i < deg; i ++){
    if (i==0){
      ans = ans + 1.0;
    } else {
      int n_fac = factorial(i);
      float denom = static_cast<float>(n_fac);
      ans = ans + pow(x, i) / denom;
      // printf("%f", ans);
      // printf("%f", denom);
    }
  }
  // printf("%f", ans);
  return ans;
}

__device__
int factorial(int n){
  int ans = 1;
  for (int i = n; i > 0; i--){
    ans = ans * i;
  }
  return ans;
}

int main(void)
{
  int N = 1<<20;
  float *x, *y;

  // Allocate Unified Memory – accessible from CPU or GPU
  hipMallocManaged(&x, N*sizeof(float));
  hipMallocManaged(&y, N*sizeof(float));

  // initialize x and y arrays on the host
  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  // Run kernel on 1M elements on the GPU
  int blockSize = 256;
  int numBlocks = std::ceil((N + blockSize - 1) / blockSize);
  add<<<numBlocks, blockSize>>>(N, x, y);

  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();

  // Check for errors (all values should be 3.0f)
  float maxError = 0.0f;
  for (int i = 0; i < N; i++)
    maxError = fmax(maxError, fabs(y[i]-7.389f));
  std::cout << "Max error: " << maxError << std::endl;

  // Free memory
  hipFree(x);
  hipFree(y);
  
  return 0;
}